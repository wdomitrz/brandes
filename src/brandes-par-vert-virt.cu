#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <cstring>
#include <iostream>

#include "brandes-virt.hpp"
#include "errors.hpp"
#include "sizes.hpp"

__global__ void brandes_kernel(const uint32_t n, const uint32_t virt_n,
                               const uint32_t starting_positions[],
                               const uint32_t compact_graph[],
                               const uint32_t vmap[], const uint32_t vptrs[],
                               double CB[], uint32_t* sigma, uint32_t* d,
                               double* delta);

void brandes(const uint32_t n, const uint32_t virt_n,
             const uint32_t starting_positions[],
             const uint32_t compact_graph[], const uint32_t vmap[],
             const uint32_t vptrs[], double CB[]) {
    if (n == 0 || starting_positions[n] == 0) return;
    uint32_t *starting_positions_dev, *compact_graph_dev, *vmap_dev, *vptrs_dev,
        *sigma, *d;
    double *delta, *CB_dev;
    HANDLE_ERROR(hipMalloc((void**)&starting_positions_dev,
                            sizeof(uint32_t) * (n + 1)));
    HANDLE_ERROR(hipMalloc((void**)&compact_graph_dev,
                            sizeof(uint32_t) * starting_positions[n]));
    HANDLE_ERROR(hipMalloc((void**)&vmap_dev, sizeof(uint32_t) * virt_n));
    HANDLE_ERROR(
        hipMalloc((void**)&vptrs_dev, sizeof(uint32_t) * (virt_n + 1)));
    HANDLE_ERROR(hipMalloc((void**)&CB_dev, sizeof(double) * n));
    HANDLE_ERROR(hipMalloc((void**)&sigma, sizeof(uint32_t) * n * BLOCKS));
    HANDLE_ERROR(hipMalloc((void**)&d, sizeof(uint32_t) * n * BLOCKS));
    HANDLE_ERROR(hipMalloc((void**)&delta, sizeof(double) * n * BLOCKS));
    HANDLE_ERROR(hipMemcpy(starting_positions_dev, starting_positions,
                            sizeof(uint32_t) * (n + 1),
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(vmap_dev, vmap, sizeof(uint32_t) * virt_n,
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(vptrs_dev, vptrs, sizeof(uint32_t) * (virt_n + 1),
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(compact_graph_dev, compact_graph,
                            sizeof(uint32_t) * starting_positions[n],
                            hipMemcpyHostToDevice));
    // HANDLE_ERROR(hipMemset(CB_res, 0.0, sizeof(double) * n));
    brandes_kernel<<<BLOCKS, THREADS>>>(n, virt_n, starting_positions_dev,
                                        compact_graph_dev, vmap_dev, vptrs_dev,
                                        CB_dev, sigma, d, delta);
    HANDLE_ERROR(
        hipMemcpy(CB, CB_dev, sizeof(double) * n, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(delta));
    HANDLE_ERROR(hipFree(d));
    HANDLE_ERROR(hipFree(sigma));
    HANDLE_ERROR(hipFree(CB_dev));
    HANDLE_ERROR(hipFree(compact_graph_dev));
    HANDLE_ERROR(hipFree(vmap_dev));
    HANDLE_ERROR(hipFree(vptrs_dev));
    HANDLE_ERROR(hipFree(starting_positions_dev));
}

__global__ void brandes_kernel(const uint32_t n, const uint32_t virt_n,
                               const uint32_t starting_positions[],
                               const uint32_t compact_graph[],
                               const uint32_t vmap[], const uint32_t vptrs[],
                               double CB[], uint32_t* sigma_global,
                               uint32_t* d_global, double* delta_global) {
    const uint32_t my_start = threadIdx.x;
    const uint32_t my_end = n;
    const uint32_t my_step = blockDim.x;
    __shared__ bool cont;
    __shared__ uint32_t l;
    __shared__ uint32_t* sigma;
    __shared__ uint32_t* d;
    __shared__ double* delta;
    if (threadIdx.x == 0) {
        sigma = &sigma_global[n * blockIdx.x];
        d = &d_global[n * blockIdx.x];
        delta = &delta_global[n * blockIdx.x];
    }
    if (blockIdx.x == 0)
        for (int i = my_start; i < my_end; i += my_step) {
            CB[i] = 0;
        }
    for (uint32_t s = blockIdx.x; s < n; s += gridDim.x) {
        __syncthreads();
        for (int i = my_start; i < my_end; i += my_step) {
            sigma[i] = 0;
            d[i] = UINT32_MAX;
            delta[i] = 1.0;
        }
        __syncthreads();
        if (my_start == 0) {
            sigma[s] = 1;
            d[s] = 0;
            cont = true;
            l = 0;
        }
        __syncthreads();
        while (cont) {
            __syncthreads();
            cont = false;
            __syncthreads();
            for (uint32_t u_virt = my_start; u_virt < virt_n;
                 u_virt += my_step) {
                const uint32_t u = vmap[u_virt];
                if (d[u] == l) {
                    const uint32_t end = vptrs[u_virt + 1];
                    for (uint32_t i = vptrs[u_virt]; i < end; i++) {
                        const uint32_t v = compact_graph[i];
                        if (d[v] == UINT32_MAX) {
                            d[v] = l + 1;
                            cont = true;
                        }
                        if (d[v] == l + 1) {
                            atomicAdd(&sigma[v], sigma[u]);
                        }
                    }
                }
            }
            __syncthreads();
            if (threadIdx.x == 0) {
                l++;
            }
        }
        __syncthreads();
        while (l > 1) {
            __syncthreads();
            if (threadIdx.x == 0) l--;
            __syncthreads();
            for (uint32_t u_virt = my_start; u_virt < virt_n;
                 u_virt += my_step) {
                const uint32_t u = vmap[u_virt];
                if (d[u] == l) {
                    double sum = 0;
                    const uint32_t end = vptrs[u_virt + 1];
                    for (uint32_t i = vptrs[u_virt]; i < end; i++) {
                        const uint32_t v = compact_graph[i];
                        if (d[v] == l + 1) {
                            sum +=
                                (double)sigma[u] / (double)sigma[v] * delta[v];
                        }
                    }
                    atomicAdd(&delta[u], sum);
                }
            }
        }
        __syncthreads();
        for (uint32_t v = my_start; v < my_end; v += my_step) {
            if (v != s) {
                atomicAdd(&CB[v], (delta[v] - (double)1));
            }
        }
    }
}
