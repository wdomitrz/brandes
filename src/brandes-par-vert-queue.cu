#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <cstring>
#include <iostream>

#include "brandes-old.hpp"
#include "errors.hpp"
#include "sizes.hpp"

__global__ void brandes_kernel(const uint32_t n,
                               const uint32_t starting_positions[],
                               const uint32_t compact_graph[], double CB[],
                               uint32_t* sigma, uint32_t* d, double* delta,
                               uint32_t* Q);

void brandes(const uint32_t n, const uint32_t starting_positions[],
             const uint32_t compact_graph[], double CB[]) {
    uint32_t *starting_positions_dev, *compact_graph_dev, *sigma, *d, *Q;
    double *delta, *CB_dev;
    HANDLE_ERROR(hipMalloc((void**)&starting_positions_dev,
                            sizeof(uint32_t) * (n + 1)));
    HANDLE_ERROR(hipMalloc((void**)&compact_graph_dev,
                            sizeof(uint32_t) * starting_positions[n]));
    HANDLE_ERROR(hipMalloc((void**)&CB_dev, sizeof(double) * n));
    HANDLE_ERROR(hipMalloc((void**)&Q, sizeof(uint32_t) * 2 * n * BLOCKS));
    HANDLE_ERROR(hipMalloc((void**)&sigma, sizeof(uint32_t) * n * BLOCKS));
    HANDLE_ERROR(hipMalloc((void**)&d, sizeof(uint32_t) * n * BLOCKS));
    HANDLE_ERROR(hipMalloc((void**)&delta, sizeof(double) * n * BLOCKS));
    HANDLE_ERROR(hipMemcpy(starting_positions_dev, starting_positions,
                            sizeof(uint32_t) * (n + 1),
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(compact_graph_dev, compact_graph,
                            sizeof(uint32_t) * starting_positions[n],
                            hipMemcpyHostToDevice));
    // HANDLE_ERROR(hipMemset(CB_res, 0.0, sizeof(double) * n));
    brandes_kernel<<<BLOCKS, THREADS>>>(n, starting_positions_dev,
                                        compact_graph_dev, CB_dev, sigma, d,
                                        delta, Q);
    HANDLE_ERROR(
        hipMemcpy(CB, CB_dev, sizeof(double) * n, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(delta));
    HANDLE_ERROR(hipFree(d));
    HANDLE_ERROR(hipFree(sigma));
    HANDLE_ERROR(hipFree(CB_dev));
    HANDLE_ERROR(hipFree(compact_graph_dev));
    HANDLE_ERROR(hipFree(starting_positions_dev));
}

__global__ void brandes_kernel(const uint32_t n,
                               const uint32_t starting_positions[],
                               const uint32_t compact_graph[], double CB[],
                               uint32_t* sigma_global, uint32_t* d_global,
                               double* delta_global, uint32_t* Q_all_global) {
    const uint32_t my_start = threadIdx.x;
    const uint32_t my_end = n;
    const uint32_t my_step = blockDim.x;
    __shared__ uint32_t l;
    __shared__ uint32_t* sigma;
    __shared__ uint32_t* d;
    __shared__ double* delta;
    __shared__ uint32_t *Q, *next_Q, *Q_global;
    __shared__ uint32_t Q_size, next_Q_size;
    if (threadIdx.x == 0) {
        sigma = &sigma_global[n * blockIdx.x];
        d = &d_global[n * blockIdx.x];
        delta = &delta_global[n * blockIdx.x];
        Q_global = &Q_all_global[2 * n * blockIdx.x];
    }
    if (blockIdx.x == 0)
        for (int i = my_start; i < my_end; i += my_step) {
            CB[i] = 0;
        }
    for (uint32_t s = blockIdx.x; s < n; s += gridDim.x) {
        __syncthreads();
        for (int i = my_start; i < my_end; i += my_step) {
            sigma[i] = 0;
            d[i] = UINT32_MAX;
            delta[i] = 0.0;
        }
        __syncthreads();
        if (my_start == 0) {
            sigma[s] = 1;
            d[s] = 0;
            l = 0;
            Q = &Q_global[n * (l % 2)];
            next_Q = &Q_global[n * ((l + 1) % 2)];
            next_Q_size = 0;
            Q_size = 1;
            Q[0] = s;
        }
        __syncthreads();
        while (Q_size != 0) {
            __syncthreads();
            for (uint32_t j = my_start; j < Q_size; j += my_step) {
                const uint32_t u = Q[j];
                const uint32_t end = starting_positions[u + 1];
                for (uint32_t i = starting_positions[u]; i < end; i++) {
                    const uint32_t v = compact_graph[i];
                    if (atomicCAS(&d[v], UINT32_MAX, l + 1) == UINT32_MAX) {
                        // add to next Q
                        next_Q[atomicAdd(&next_Q_size, 1)] = v;
                    }
                    if (d[v] == l + 1) {
                        atomicAdd(&sigma[v], sigma[u]);
                    }
                }
            }
            __syncthreads();
            if (threadIdx.x == 0) {
                l++;
                Q = &Q_global[n * (l % 2)];
                next_Q = &Q_global[n * ((l + 1) % 2)];
                Q_size = next_Q_size;
                next_Q_size = 0;
            }
            __syncthreads();
        }
        __syncthreads();
        while (l > 1) {
            __syncthreads();
            if (threadIdx.x == 0) l--;
            __syncthreads();
            for (uint32_t u = my_start; u < my_end; u += my_step) {
                if (d[u] == l) {
                    const uint32_t end = starting_positions[u + 1];
                    for (uint32_t i = starting_positions[u]; i < end; i++) {
                        const uint32_t v = compact_graph[i];
                        if (d[v] - 1 == d[u]) {
                            delta[u] += ((double)sigma[u]) /
                                        ((double)sigma[v]) *
                                        ((double)1.0 + (double)delta[v]);
                        }
                    }
                }
            }
        }
        __syncthreads();
        for (uint32_t v = my_start; v < my_end; v += my_step) {
            if (v != s) {
                atomicAdd(&CB[v], delta[v]);
            }
        }
    }
}
