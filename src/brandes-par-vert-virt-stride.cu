#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <cstring>
#include <iostream>

#include "brandes-virt-stride-nocomp.hpp"
#include "errors.hpp"
#include "sizes.hpp"

#define allign_up_to_ALLIGN
#define allign_up_to_ALLIGN_dev
// #define ALLIGN 32

// inline size_t allign_up_to_ALLIGN_dev(size_t x) {
//     if (x % ALLIGN == 0)
//         return x;
//     else
//         return (x + (x - (x % ALLIGN)));
// }

// __device__ inline size_t allign_up_to_ALLIGN_dev(size_t x) {
//     if (x % ALLIGN == 0)
//         return x;
//     else
//         return (x + (x - (x % ALLIGN)));
// }

__global__ void brandes_kernel(const int32_t n, const int32_t virt_n,
                               const int32_t starting_positions[],
                               const int32_t compact_graph[],
                               const int32_t vmap[], const int32_t vptrs[],
                               const int32_t jmp[], double CB[], int32_t* sigma,
                               int32_t* d, double* delta);

void brandes(const int32_t n, const int32_t virt_n,
             const int32_t starting_positions[], const int32_t compact_graph[],
             const int32_t vmap[], const int32_t vptrs[], const int32_t jmp[],
             double CB[]) {
    if (n == 0 || starting_positions[n] == 0) return;
    int32_t *starting_positions_dev, *compact_graph_dev, *vmap_dev, *vptrs_dev,
        *jmp_dev, *sigma, *d;
    double *delta, *CB_dev;
    HANDLE_ERROR(
        hipMalloc((void**)&starting_positions_dev, sizeof(int32_t) * (n + 1)));
    HANDLE_ERROR(hipMalloc((void**)&jmp_dev, sizeof(int32_t) * n));
    HANDLE_ERROR(hipMalloc((void**)&compact_graph_dev,
                            sizeof(int32_t) * starting_positions[n]));
    HANDLE_ERROR(hipMalloc((void**)&vmap_dev, sizeof(int32_t) * virt_n));
    HANDLE_ERROR(
        hipMalloc((void**)&vptrs_dev, sizeof(int32_t) * (virt_n + 1)));
    HANDLE_ERROR(hipMalloc((void**)&CB_dev, sizeof(double) * n));
    HANDLE_ERROR(hipMalloc((void**)&sigma, sizeof(int32_t) * n * BLOCKS));
    HANDLE_ERROR(hipMalloc((void**)&d, sizeof(int32_t) * n * BLOCKS));
    HANDLE_ERROR(hipMalloc((void**)&delta, sizeof(double) * n * BLOCKS));
    HANDLE_ERROR(hipMemcpy(starting_positions_dev, starting_positions,
                            sizeof(int32_t) * (n + 1), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(vmap_dev, vmap, sizeof(int32_t) * virt_n,
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(vptrs_dev, vptrs, sizeof(int32_t) * (virt_n + 1),
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(
        hipMemcpy(jmp_dev, jmp, sizeof(int32_t) * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(compact_graph_dev, compact_graph,
                            sizeof(int32_t) * starting_positions[n],
                            hipMemcpyHostToDevice));
    // HANDLE_ERROR(hipMemset(CB_res, 0.0, sizeof(double) * n));
    brandes_kernel<<<BLOCKS, THREADS>>>(n, virt_n, starting_positions_dev,
                                        compact_graph_dev, vmap_dev, vptrs_dev,
                                        jmp_dev, CB_dev, sigma, d, delta);
    HANDLE_ERROR(
        hipMemcpy(CB, CB_dev, sizeof(double) * n, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(delta));
    HANDLE_ERROR(hipFree(d));
    HANDLE_ERROR(hipFree(sigma));
    HANDLE_ERROR(hipFree(CB_dev));
    HANDLE_ERROR(hipFree(compact_graph_dev));
    HANDLE_ERROR(hipFree(vmap_dev));
    HANDLE_ERROR(hipFree(vptrs_dev));
    HANDLE_ERROR(hipFree(jmp_dev));
    HANDLE_ERROR(hipFree(starting_positions_dev));
}

__global__ void brandes_kernel(const int32_t n, const int32_t virt_n,
                               const int32_t starting_positions[],
                               const int32_t compact_graph[],
                               const int32_t vmap[], const int32_t vptrs[],
                               const int32_t jmp[], double CB[],
                               int32_t* sigma_global, int32_t* d_global,
                               double* delta_global) {
    // const int32_t big_step = 1 + (n - 1) / blockDim.x;
    // const int32_t my_start = threadIdx. * big_step;
    // const int32_t my_end = min(n, (threadIdx.x + 1) * big_step);
    // const int32_t my_step = 1;
    const int32_t my_start = threadIdx.x;
    const int32_t my_end = n;
    const int32_t my_end_virt = virt_n;
    const int32_t my_step = blockDim.x;
    __shared__ bool cont;
    __shared__ int32_t l;
    __shared__ int32_t* sigma;
    __shared__ int32_t* d;
    __shared__ double* delta;
    if (my_start == 0) {
        sigma = &sigma_global[n * blockIdx.x];
        d = &d_global[n * blockIdx.x];
        delta = &delta_global[n * blockIdx.x];
    }
    if (blockIdx.x == 0)
        for (int i = my_start; i < my_end; i += my_step) {
            CB[i] = 0;
        }
    for (int32_t s = blockIdx.x; s < my_end; s += gridDim.x) {
        __syncthreads();
        for (int i = my_start; i < my_end; i += my_step) {
            sigma[i] = 0;
            d[i] = -1;
            delta[i] = 1.0;
        }
        __syncthreads();
        if (my_start == 0) {
            sigma[s] = 1;
            d[s] = 0;
            cont = true;
            l = 0;
        }
        __syncthreads();
        while (cont) {
            __syncthreads();
            cont = false;
            __syncthreads();
            for (int32_t u_virt = my_start; u_virt < my_end_virt;
                 u_virt += my_step) {
                const int32_t u = vmap[u_virt];
                if (d[u] == l) {
                    const int32_t end = starting_positions[u + 1];
                    const int32_t now_jmp = jmp[u];
                    for (int32_t i = vptrs[u_virt]; i < end; i += now_jmp) {
                        const int32_t v = compact_graph[i];
                        if (d[v] == -1) {
                            d[v] = l + 1;
                            cont = true;
                        }
                        if (d[v] == l + 1) {
                            atomicAdd(&sigma[v], sigma[u]);
                        }
                    }
                }
            }
            __syncthreads();
            if (my_start == 0) {
                l++;
            }
        }
        __syncthreads();
        while (l > 1) {
            __syncthreads();
            if (my_start == 0) l--;
            __syncthreads();
            for (int32_t u_virt = my_start; u_virt < my_end_virt;
                 u_virt += my_step) {
                const int32_t u = vmap[u_virt];
                if (d[u] == l) {
                    double sum = 0;
                    const int32_t end = starting_positions[u + 1];
                    const int32_t now_jmp = jmp[u];
                    for (int32_t i = vptrs[u_virt]; i < end; i += now_jmp) {
                        const int32_t v = compact_graph[i];
                        if (d[v] == l + 1) {
                            sum +=
                                (double)sigma[u] / (double)sigma[v] * delta[v];
                        }
                    }
                    atomicAdd(&delta[u], sum);
                }
            }
        }
        __syncthreads();
        for (int32_t v = my_start; v < my_end; v += my_step) {
            if (v != s) {
                atomicAdd(&CB[v], (delta[v] - (double)1));
            }
        }
    }
}
