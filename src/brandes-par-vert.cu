#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <cstring>
#include <iostream>

#include "brandes.hpp"
#include "errors.hpp"
#define add_to_S(x) \
    { S[S_size++] = x; }
#define add_to_P(x, y) \
    { P[starting_positions[x] + P_pos[x]++] = y; }

__global__ void brandes_kernel(const int32_t n,
                               const int32_t starting_positions[],
                               const int32_t compact_graph[], double CB[],
                               int32_t* sigma, int32_t* d, double* delta,
                               int32_t* P, int32_t* P_pos);

void brandes(const int32_t n, const int32_t starting_positions[],
             const int32_t compact_graph[], double CB[]) {
    int32_t *starting_positions_dev, *compact_graph_dev, *sigma, *d, *P, *P_pos;
    double *delta, *CB_dev;
    HANDLE_ERROR(
        hipMalloc((void**)&starting_positions_dev, sizeof(int32_t) * (n + 1)));
    HANDLE_ERROR(hipMalloc((void**)&compact_graph_dev,
                            sizeof(int32_t) * starting_positions[n]));
    HANDLE_ERROR(hipMalloc((void**)&CB_dev, sizeof(double) * n));
    HANDLE_ERROR(hipMalloc((void**)&sigma, sizeof(int32_t) * n));
    HANDLE_ERROR(hipMalloc((void**)&d, sizeof(int32_t) * n));
    HANDLE_ERROR(hipMalloc((void**)&delta, sizeof(double) * n));
    HANDLE_ERROR(
        hipMalloc((void**)&P, sizeof(int32_t) * starting_positions[n]));
    HANDLE_ERROR(hipMalloc((void**)&P_pos, sizeof(int32_t) * n));
    HANDLE_ERROR(hipMemcpy(starting_positions_dev, starting_positions,
                            sizeof(int32_t) * (n + 1), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(compact_graph_dev, compact_graph,
                            sizeof(int32_t) * starting_positions[n],
                            hipMemcpyHostToDevice));
    // HANDLE_ERROR(hipMemset(CB_dev, 0.0, sizeof(double) * n));
    brandes_kernel<<<1, 1>>>(n, starting_positions_dev, compact_graph_dev,
                             CB_dev, sigma, d, delta, P, P_pos);
    HANDLE_ERROR(
        hipMemcpy(CB, CB_dev, sizeof(double) * n, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(P_pos));
    HANDLE_ERROR(hipFree(P));
    HANDLE_ERROR(hipFree(delta));
    HANDLE_ERROR(hipFree(d));
    HANDLE_ERROR(hipFree(sigma));
    HANDLE_ERROR(hipFree(CB_dev));
    HANDLE_ERROR(hipFree(compact_graph_dev));
    HANDLE_ERROR(hipFree(starting_positions_dev));
}

__global__ void brandes_kernel(const int32_t n,
                               const int32_t starting_positions[],
                               const int32_t compact_graph[], double CB[],
                               int32_t* sigma, int32_t* d, double* delta,
                               int32_t* P, int32_t* P_pos) {
    const int32_t my_start = threadIdx.x + blockIdx.x * blockDim.x;
    const int32_t my_end = n;
    const int32_t my_step = blockDim.x * gridDim.x;
    __shared__ bool cont;
    __shared__ int32_t l;
    for (int i = my_start; i < my_end; i += my_step) {
        CB[i] = 0;
    }
    __syncthreads();
    for (int32_t s = 0; s < n; s++) {
        for (int i = my_start; i < my_end; i += my_step) {
            sigma[i] = 0;
            d[i] = -1;
            delta[i] = 0.0;
            P_pos[i] = 0;
        }
        __syncthreads();
        if (my_start == 0) {
            sigma[s] = 1;
            d[s] = 0;
            cont = true;
            l = 0;
        }
        __syncthreads();
        for (; cont; l++) {
            cont = false;
            __syncthreads();
            for (int32_t u = my_start; u < my_end; u += my_step) {
                if (d[u] == l) {
                    const int32_t end = starting_positions[u + 1];
                    for (int32_t i = starting_positions[u]; i < end; i++) {
                        const int32_t v = compact_graph[i];
                        if (d[v] == -1) {
                            d[v] = l + 1;
                            cont = true;
                        } else if (d[v] == l - 1) {
                            add_to_P(v, u);
                        }
                        if (d[v] == l + 1) {
                            atomicAdd(&sigma[v], sigma[u]);
                        }
                    }
                }
            }
        }
        for (; l > 1;) {
            l--;
            __syncthreads();
            for (int32_t u = my_start; u < my_end; u += my_step) {
                if (d[u] == l) {
                    int32_t P_iter_end = starting_positions[u] + P_pos[u];
                    for (int32_t i = starting_positions[u]; i < P_iter_end;
                         i++) {
                        const int32_t v = P[i];
                        if (sigma[u] != 0) {
                            delta[u] += ((double)sigma[u]) /
                                        ((double)sigma[v]) *
                                        ((double)1.0 + (double)delta[v]);
                        }
                    }
                }
            }
        }
        __syncthreads();
        for (int32_t v = my_start; v < my_end; v += my_step) {
            if (v != s) {
                CB[v] += delta[v];
            }
        }
    }
}
