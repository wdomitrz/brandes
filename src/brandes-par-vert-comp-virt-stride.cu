#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <cstring>
#include <iostream>

#include "brandes-virt-stride.hpp"
#include "errors.hpp"
#include "sizes.hpp"

#define allign_up_to_ALLIGN
#define allign_up_to_ALLIGN_dev
// #define ALLIGN 32

// inline size_t allign_up_to_ALLIGN_dev(size_t x) {
//     if (x % ALLIGN == 0)
//         return x;
//     else
//         return (x + (x - (x % ALLIGN)));
// }

// __device__ inline size_t allign_up_to_ALLIGN_dev(size_t x) {
//     if (x % ALLIGN == 0)
//         return x;
//     else
//         return (x + (x - (x % ALLIGN)));
// }

__global__ void brandes_kernel(const uint32_t n, const uint32_t virt_n,
                               const uint32_t starting_positions[],
                               const uint32_t compact_graph[],
                               const uint32_t reach[], const uint32_t vmap[],
                               const uint32_t vptrs[], const uint32_t jmp[],
                               double CB[], uint32_t* sigma, uint32_t* d,
                               double* delta);

__global__ void collect_CB(const size_t n, double CB[], const size_t end) {
    for (size_t which = blockIdx.x * blockDim.x + threadIdx.x; which < n;
         which += blockDim.x * gridDim.x) {
        for (size_t i = 1; i < end; i++) {
            CB[which] += CB[which + n * i];
        }
    }
}

void brandes(const uint32_t n, const uint32_t virt_n,
             const uint32_t starting_positions[],
             const uint32_t compact_graph[], const uint32_t reach[],
             const uint32_t vmap[], const uint32_t vptrs[],
             const uint32_t jmp[], double CB[]) {
    if (n == 0 || starting_positions[n] == 0) {
        std::cerr << 0 << "\n" << 0 << "\n";
        return;
    }
    uint32_t *starting_positions_dev, *reach_dev, *compact_graph_dev, *vmap_dev,
        *vptrs_dev, *jmp_dev, *d;
    uint32_t* sigma;
    double *delta, *CB_dev;
    hipStream_t stream[6];
    for (size_t i = 0; i < 6; i++) {
        hipStreamCreate(&stream[i]);
    }
    hipEvent_t start_kernel, stop_kernel, start_with_memory, stop_with_memory;
    HANDLE_ERROR(hipEventCreate(&start_with_memory));
    HANDLE_ERROR(hipEventCreate(&start_kernel));
    HANDLE_ERROR(hipEventCreate(&stop_kernel));
    HANDLE_ERROR(hipEventCreate(&stop_with_memory));
    HANDLE_ERROR(hipMalloc((void**)&starting_positions_dev,
                            sizeof(uint32_t) * (n + 1)));
    HANDLE_ERROR(hipMalloc((void**)&reach_dev, sizeof(uint32_t) * n));
    HANDLE_ERROR(hipMalloc((void**)&jmp_dev, sizeof(uint32_t) * n));
    HANDLE_ERROR(hipMalloc((void**)&compact_graph_dev,
                            sizeof(uint32_t) * starting_positions[n]));
    HANDLE_ERROR(hipMalloc((void**)&vmap_dev, sizeof(uint32_t) * virt_n));
    HANDLE_ERROR(
        hipMalloc((void**)&vptrs_dev, sizeof(uint32_t) * (virt_n + 1)));
    HANDLE_ERROR(hipMalloc((void**)&CB_dev, sizeof(double) * n * BLOCKS));
    HANDLE_ERROR(hipMalloc((void**)&sigma, sizeof(uint32_t) * n * BLOCKS));
    HANDLE_ERROR(hipMalloc((void**)&d, sizeof(uint32_t) * n * BLOCKS));
    HANDLE_ERROR(hipMalloc((void**)&delta, sizeof(double) * n * BLOCKS));
    HANDLE_ERROR(hipEventRecord(start_with_memory, 0));
    HANDLE_ERROR(hipMemcpyAsync(starting_positions_dev, starting_positions,
                                 sizeof(uint32_t) * (n + 1),
                                 hipMemcpyHostToDevice, stream[0]));
    HANDLE_ERROR(hipMemcpyAsync(reach_dev, reach, sizeof(uint32_t) * n,
                                 hipMemcpyHostToDevice, stream[1]));
    HANDLE_ERROR(hipMemcpyAsync(vmap_dev, vmap, sizeof(uint32_t) * virt_n,
                                 hipMemcpyHostToDevice, stream[2]));
    HANDLE_ERROR(hipMemcpyAsync(vptrs_dev, vptrs,
                                 sizeof(uint32_t) * (virt_n + 1),
                                 hipMemcpyHostToDevice, stream[3]));
    HANDLE_ERROR(hipMemcpyAsync(jmp_dev, jmp, sizeof(uint32_t) * n,
                                 hipMemcpyHostToDevice, stream[4]));
    HANDLE_ERROR(hipMemcpyAsync(compact_graph_dev, compact_graph,
                                 sizeof(uint32_t) * starting_positions[n],
                                 hipMemcpyHostToDevice, stream[5]));
    HANDLE_ERROR(hipEventRecord(start_kernel, 0));
    brandes_kernel<<<BLOCKS, THREADS, 0, 0>>>(
        n, virt_n, starting_positions_dev, compact_graph_dev, reach_dev,
        vmap_dev, vptrs_dev, jmp_dev, CB_dev, sigma, d, delta);
    HANDLE_ERROR(hipEventRecord(stop_kernel, 0));
    HANDLE_ERROR(hipEventSynchronize(stop_kernel));
    collect_CB<<<BLOCKS, THREADS, 0, 0>>>(n, CB_dev, BLOCKS);
    HANDLE_ERROR(
        hipMemcpy(CB, CB_dev, sizeof(double) * n, hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipEventRecord(stop_with_memory, 0));
    HANDLE_ERROR(hipEventSynchronize(stop_with_memory));

    float time_kernel, time_with_memory;
    HANDLE_ERROR(hipEventElapsedTime(&time_kernel, start_kernel, stop_kernel));
    HANDLE_ERROR(hipEventElapsedTime(&time_with_memory, start_with_memory,
                                      stop_with_memory));

    std::cerr << (unsigned long long)time_kernel << "\n"
              << (unsigned long long)time_with_memory << "\n";

    HANDLE_ERROR(hipEventDestroy(start_with_memory));
    HANDLE_ERROR(hipEventDestroy(start_kernel));
    HANDLE_ERROR(hipEventDestroy(stop_kernel));
    HANDLE_ERROR(hipEventDestroy(stop_with_memory));

    HANDLE_ERROR(hipFree(delta));
    HANDLE_ERROR(hipFree(d));
    HANDLE_ERROR(hipFree(sigma));
    HANDLE_ERROR(hipFree(CB_dev));
    HANDLE_ERROR(hipFree(compact_graph_dev));
    HANDLE_ERROR(hipFree(reach_dev));
    HANDLE_ERROR(hipFree(vmap_dev));
    HANDLE_ERROR(hipFree(vptrs_dev));
    HANDLE_ERROR(hipFree(jmp_dev));
    HANDLE_ERROR(hipFree(starting_positions_dev));
}

__global__ void brandes_kernel(const uint32_t n, const uint32_t virt_n,
                               const uint32_t starting_positions[],
                               const uint32_t compact_graph[],
                               const uint32_t reach[], const uint32_t vmap[],
                               const uint32_t vptrs[], const uint32_t jmp[],
                               double CB_global[], uint32_t* sigma_global,
                               uint32_t* d_global, double* delta_global) {
    // const uint32_t big_step = 1 + (n - 1) / blockDim.x;
    // const uint32_t my_start = threadIdx. * big_step;
    // const uint32_t my_end = min(n, (threadIdx.x + 1) * big_step);
    // const uint32_t my_step = 1;
    const uint32_t my_start = threadIdx.x;
    const uint32_t my_end = n;
    const uint32_t my_end_virt = virt_n;
    const uint32_t my_step = blockDim.x;
    __shared__ bool cont;
    __shared__ uint32_t l;
    __shared__ uint32_t* sigma;
    __shared__ uint32_t* d;
    __shared__ double* delta;
    __shared__ double* CB;
    if (my_start == 0) {
        sigma = &sigma_global[n * blockIdx.x];
        d = &d_global[n * blockIdx.x];
        delta = &delta_global[n * blockIdx.x];
        CB = &CB_global[n * blockIdx.x];
    }
    __syncthreads();
    for (uint32_t i = my_start; i < my_end; i += my_step) {
        CB[i] = 0;
    }
    for (uint32_t s = blockIdx.x; s < my_end; s += gridDim.x) {
        __syncthreads();
        for (uint32_t i = my_start; i < my_end; i += my_step) {
            sigma[i] = 0;
            d[i] = UINT32_MAX;
            delta[i] = reach[i];
        }
        __syncthreads();
        if (my_start == 0) {
            sigma[s] = 1;
            d[s] = 0;
            cont = true;
            l = 0;
        }
        __syncthreads();
        while (cont) {
            __syncthreads();
            cont = false;
            __syncthreads();
            for (uint32_t u_virt = my_start; u_virt < my_end_virt;
                 u_virt += my_step) {
                const uint32_t u = vmap[u_virt];
                if (d[u] == l) {
                    const uint32_t end = starting_positions[u + 1];
                    const uint32_t now_jmp = jmp[u];
                    for (uint32_t i = vptrs[u_virt]; i < end; i += now_jmp) {
                        const uint32_t v = compact_graph[i];
                        if (d[v] == UINT32_MAX) {
                            d[v] = l + 1;
                            cont = true;
                        }
                        if (d[v] == l + 1) {
                            atomicAdd(&sigma[v], sigma[u]);
                        }
                    }
                }
            }
            __syncthreads();
            if (my_start == 0) {
                l++;
            }
        }
        __syncthreads();
        while (l > 1) {
            __syncthreads();
            if (my_start == 0) l--;
            __syncthreads();
            for (uint32_t u_virt = my_start; u_virt < my_end_virt;
                 u_virt += my_step) {
                const uint32_t u = vmap[u_virt];
                if (d[u] == l) {
                    double sum = 0;
                    const uint32_t end = starting_positions[u + 1];
                    const uint32_t now_jmp = jmp[u];
                    for (uint32_t i = vptrs[u_virt]; i < end; i += now_jmp) {
                        const uint32_t v = compact_graph[i];
                        if (d[v] == l + 1) {
                            sum +=
                                (double)sigma[u] / (double)sigma[v] * delta[v];
                        }
                    }
                    atomicAdd(&delta[u], sum);
                }
            }
        }
        __syncthreads();
        for (uint32_t v = my_start; v < my_end; v += my_step) {
            if (v != s) {
                CB[v] += (double)reach[s] * (delta[v] - (double)reach[v]);
            }
        }
    }
}
